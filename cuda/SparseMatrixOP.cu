#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

using namespace std;

class SM{
public:
    
    //insertion and sort helper functions
    template<typename T>
    __host__ void insertElement(thrust::host_vector<T> &vect,int idx, T value){
        
        //First increase the size by 1
        vect.push_back(0);
        
        T temp = vect[idx];
        vect[idx] = value;
        for(int i=idx+1;i<vect.size();i++){
            T temp_ = vect[i];
            vect[i] = temp;
            temp = temp_;
        }
        return;
    }
    
    template<typename T>
    __host__ void sortElements(thrust::host_vector<T> &vect, int startIdx, int endIdx){
    
        for(int i=startIdx;i<endIdx-1;i++){
            for(int j=startIdx+1;j<endIdx;j++){
                if(vect[i]>vect[j]){
                    T temp = vect[i];
                    vect[i] = vect[j];
                    vect[j] = temp;
                }
            }
        }
        return;
    }
    
public:
    int row, col, size;
    
    thrust::host_vector<double> val;
    thrust::host_vector<int> column_index;
    thrust::host_vector<int> row_index;
    thrust::host_vector<int> rows;
    
    __host__ SM(){
        this->row = 0;
        this->col = 0;
        this->size = 0;
    }
    
    __host__ void insert(int r, int c, double v){
        if(v==0){
         printf("Input value 0 is ignored in the sparse matrix operation...\n");
         return;
        }
        
        /*initial case*/
        if(this->size==0){
            row_index.push_back(0);
            rows.push_back(r);
            column_index.push_back(c);
            val.push_back(v);
            this->row = r+1;
            this->col = c+1;
            this->size++;
            return;
        }
        
        /*rest*/
        bool updated = false;
        if(r>=this->row){this->row = r+1;}
        if(c>=this->col){this->col = c+1;}
        for(int i=0;i<rows.size();i++){
            //if updated in the previous loop, the remainder row_index needs to be shifted by 1, and continue looping till the end
            if(updated){
                row_index[i] += 1;
                continue;
            }
        
            //In case the row already exists
            if(r==rows[i]){
                int N;
                if(i==rows.size()-1){
                    N = this->size - row_index[i];
                }else{
                    N = row_index[i+1] - row_index[i];
                }
                for(int j=row_index[i];j<row_index[i]+N;j++){
                    if(c==column_index[j]){
                        printf("An element already exists at the row %d and column %d\n",r,c);
                        printf("Insertion of %lf at (%d, %d) failed...\n",v,r,c);
                        return;
                    }
                }
                //Inserting a new col value in the column_index vector at the 1st position of the corresponding row lump.
                insertElement(column_index,row_index[i], c);
                sortElements(column_index, row_index[i], row_index[i]+N+1);
                int idx = row_index[i];
                for(int j=row_index[i];j<row_index[i]+N+1;j++){
                    if(c==column_index[j]){
                        idx = j;
                        break;
                    }
                }
                insertElement(val,idx,v);
                updated = true;

            }else if(r<rows[i]){//In case the row doesn't exist
                int idx = row_index[i];
                insertElement(column_index,idx,c);
                insertElement(val,idx,v);
                insertElement(row_index,i,idx);
                insertElement(rows,i,r);
                updated = true;
           
            }
        }
        
        if(!updated){
            column_index.push_back(c);
            val.push_back(v);
            rows.push_back(r);
            row_index.push_back(this->size-1);
        }

        this->size++;
        return;
    }
  
    __host__ void printSparse(){
        cout<<"Sparse matrix: "<<endl;
        for(int i=0;i<rows.size();i++){
            int N;
            if(i==rows.size()-1){
                N = this->size - row_index[i];
            }else{
                N = row_index[i+1] - row_index[i];
            }
            for(int j=0;j<this->col;j++){
                bool mark_col = false;
                int idx = 0;
                for(int k=row_index[i];k<row_index[i]+N;k++){
                    if(j==column_index[k]){
                        idx = k;
                        mark_col = true;
                        break;
                    }
                }
                if(mark_col){
                    cout<<val[idx]<<"       ";
                }else{
                    cout<<"0.0       ";
                }
            }
            cout<<endl;
        }
    }
    
    __host__ void printDense(){
        cout<<"Dense matrix: "<<endl;
        for(int i=0;i<this->row;i++){
            bool mark_row = false;
            int row_idx = 0;
            for(int r=0;r<rows.size();r++){
                if(i==rows[r]){
                    mark_row = true;
                    row_idx = r;
                    break;
                }
            }
            if(mark_row){
                int N;
                if(rows[row_idx]==this->row-1){
                    N = this->size - row_index[row_idx];
                }else{
                    N = row_index[row_idx+1] - row_index[row_idx];
                }
                for(int j=0;j<this->col;j++){
                    bool mark_col = false;
                    int idx =0;
                    for(int k=row_index[row_idx];k<row_index[row_idx]+N;k++){
                        if(j==column_index[k]){
                            idx = k;
                            mark_col = true;
                            break;
                        }
                    }
                    if(mark_col){
                        cout<<val[idx]<<"       ";
                    }else{
                        cout<<"0.0       ";
                    }
                }
                
            }else{
                for(int j=0;j<this->col;j++){
                    cout<<"0.0       ";
                }
            }
            cout<<endl;
        }
    }
 
    
};

template<class T>
void vect2arr(thrust::host_vector<T> vect,T *arr){
   for(int i=0;i<vect.size();i++){
      arr[i] = vect[i];
  }
}

__global__ void vec_product(int sparse_row_size_a, int sparse_row_size_b, double *val_a_d,double *val_b_d,
int *row_index_a_d,int *rows_a_d,int *row_index_b_d,int *rows_b_d,int *column_index_a_d,int *column_index_b_d,SM a,SM b,double *result){
    //operation validity check
    if(a.col!=b.row){
        printf("The first matrix's column dimension and the second one's row dimension doesn't match\n");
        printf("Failed to perform a product operation\n");
        return;
    }
    
    //result dim = a.row by b.col
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if(i<a.row && j<b.col){
        int idx_st_a, N_a;
        for(int r=0;r<sparse_row_size_a;r++){
            if(rows_a_d[r] == i){
                idx_st_a = row_index_a_d[r];
                if(r==sparse_row_size_a-1){
                    N_a = a.size - idx_st_a;
                }else{
                    N_a = row_index_a_d[r+1] - idx_st_a;
                }
                for(int col_a_idx = idx_st_a;col_a_idx<idx_st_a+N_a;col_a_idx++){
                    for(int l=0;l<sparse_row_size_b;l++){
                        if(column_index_a_d[col_a_idx] == rows_b_d[l]){
                            int idx_st_b, N_b;
                            idx_st_b = row_index_b_d[l];
                            if(l==sparse_row_size_b-1){
                                N_b = b.size - idx_st_b;
                            }else{
                                N_b = row_index_b_d[l+1] - idx_st_b;
                            }
                            for(int col_b_idx = idx_st_b;col_b_idx<idx_st_b+N_b;col_b_idx++){
                                if(column_index_b_d[col_b_idx]==j){
                                    //printf("(%d, %d),%lf,%lf\n",i,j,val_a_d[col_a_idx],val_b_d[col_b_idx]);
                                    result[j+b.col*i] += val_a_d[col_a_idx]*val_b_d[col_b_idx];
                                }
                            }
                        }
                    }
                }
            }
        }
        


    }
    return;
}

int main()
{
  SM a1,a2,a3;
  a1.insert(3,2,1.8);
  a1.insert(1,1,1);
  a1.insert(1,2,1.5);
  a1.insert(3,3,3.3);
  a1.insert(0,2,1.1);
  a1.insert(3,0,1);
  a1.insert(0,0,7);
  a1.insert(1,0,3);
  cout<<"----a1------"<<endl;
  a1.printSparse();
  a1.printDense();
  cout<<"__________________"<<endl;
  a2 = a1;
  cout<<"----a2------"<<endl;
  a2.printSparse();
  a2.printDense();
  cout<<"__________________"<<endl;
  
  
  //host_vector to array transformation. device_vector is only accessible on host, and on device operation is not viable.
  //Still need to figure out how best to perform operation without transforming it like this.
  //____________________________________________
  double val_a[a1.val.size()];
  int column_index_a[a1.column_index.size()];
  int row_index_a[a1.row_index.size()];
  int rows_a[a1.rows.size()];
  
  double val_b[a2.val.size()];
  int column_index_b[a2.column_index.size()];
  int row_index_b[a2.row_index.size()];
  int rows_b[a2.rows.size()];
  
  vect2arr(a1.val,val_a);
  vect2arr(a1.column_index,column_index_a);
  vect2arr(a1.row_index,row_index_a);
  vect2arr(a1.rows,rows_a);
  
  vect2arr(a2.val,val_b);
  vect2arr(a2.column_index,column_index_b);
  vect2arr(a2.row_index,row_index_b);
  vect2arr(a2.rows,rows_b);

  double *val_a_d, *val_b_d;
  int *row_index_a_d, *rows_a_d,*column_index_a_d, *column_index_b_d, *row_index_b_d, *rows_b_d;
  
  hipMalloc((void**)&val_a_d,sizeof(double)*a1.val.size());
  hipMalloc((void**)&val_b_d,sizeof(double)*a2.val.size());
  hipMalloc((void**)&row_index_a_d,sizeof(int)*a1.row_index.size());
  hipMalloc((void**)&rows_a_d,sizeof(int)*a1.rows.size());
  hipMalloc((void**)&column_index_a_d,sizeof(int)*a1.column_index.size());
  hipMalloc((void**)&column_index_b_d,sizeof(int)*a2.column_index.size());
  hipMalloc((void**)&row_index_b_d,sizeof(int)*a2.row_index.size());
  hipMalloc((void**)&rows_b_d,sizeof(int)*a2.rows.size());
  
  
  hipMemcpy(val_a_d,val_a,sizeof(double)*a1.val.size(),hipMemcpyHostToDevice);
  hipMemcpy(val_b_d,val_b,sizeof(double)*a2.val.size(),hipMemcpyHostToDevice);
  hipMemcpy(column_index_a_d,column_index_a,sizeof(int)*a1.column_index.size(),hipMemcpyHostToDevice);
  hipMemcpy(column_index_b_d,column_index_b,sizeof(int)*a2.column_index.size(),hipMemcpyHostToDevice);
  hipMemcpy(row_index_a_d,row_index_a,sizeof(int)*a1.row_index.size(),hipMemcpyHostToDevice);
  hipMemcpy(rows_a_d,rows_a,sizeof(int)*a1.rows.size(),hipMemcpyHostToDevice);
  hipMemcpy(row_index_b_d,row_index_b,sizeof(int)*a2.row_index.size(),hipMemcpyHostToDevice);
  hipMemcpy(rows_b_d,rows_b,sizeof(int)*a2.rows.size(),hipMemcpyHostToDevice);
  
  double result[a1.row*a2.col] = {0.0};
  double *result_d;
  hipMalloc((void**)&result_d,sizeof(double)*a1.row*a2.col);
  hipMemcpy(result_d,result,sizeof(double)*a1.row*a2.col,hipMemcpyHostToDevice);

  //____________________________________________
  dim3 threadsPerBlock(a1.row, a1.col);
  dim3 numBlocks((a1.row + threadsPerBlock.x -1) / threadsPerBlock.x, (a2.col +threadsPerBlock.y -1) / threadsPerBlock.y);
  
  vec_product<<<numBlocks, threadsPerBlock>>>(a1.rows.size(), a2.rows.size(),val_a_d,val_b_d,row_index_a_d,rows_a_d,row_index_b_d,rows_b_d,column_index_a_d,column_index_b_d,a1,a2,result_d);
  
  hipDeviceSynchronize();
  hipMemcpy(result, result_d, sizeof(double)*a1.row*a2.col, hipMemcpyDeviceToHost);
  printf("Product of a1 and a2 calculated!\n");
  
  for(int i=0;i<a1.row;i++){
      for(int j=0;j<a2.col;j++){
          cout<<result[j+a2.col*i]<<"       ";
      }
      cout<<endl;
  }
  
  hipFree(val_a_d);
  hipFree(val_b_d);
  hipFree(row_index_a_d);
  hipFree(rows_a_d);
  hipFree(row_index_b_d);
  hipFree(rows_b_d);
  hipFree(column_index_a_d);
  hipFree(column_index_b_d);
  hipFree(result_d);

}
